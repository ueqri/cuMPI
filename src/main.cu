#include "cuMPI_runtime.h"

int myRank;                 // cuMPI comm local ranks
int nRanks;                 // total cuMPI comm ranks
int localRank;              // CUDA device ID

ncclUniqueId id;            // NCCL Unique ID
cuMPI_Comm comm;            // cuMPI comm
hipStream_t defaultStream; // CUDA stream generated for each GPU
uint64_t hostHashs[10];     // host name hash in cuMPI
char hostname[1024];        // host name for identification in cuMPI

// test Bcast method
int main() {
  cuMPI_Init(NULL, NULL);
  
  int count = 50;
  float *h_send = (float *)malloc(count * sizeof(float)),
        *h_recv = (float *)malloc(count * sizeof(float));
  if (myRank == 0) {
    for (int i = 0; i < count; ++i) {
      h_send[i] = 2 * i + myRank;
    }
  }

  float *d_send = NULL, *d_recv = NULL;
  CUDA_CHECK(hipMalloc(&d_send, count * sizeof(float)));
  CUDA_CHECK(hipMalloc(&d_recv, count * sizeof(float)));
  
  CUDA_CHECK(hipMemcpy(d_send, h_send, count * sizeof(float), hipMemcpyHostToDevice));

  cuMPI_Bcast(d_send, count, cuMPI_FLOAT, 0, comm);

  CUDA_CHECK(hipMemcpy(h_recv, d_send, count * sizeof(float), hipMemcpyDeviceToHost));
  
  printf("[%d]:\n", myRank);
  for (int i = 0; i < count; ++i) {
    printf("%d ", (int)h_recv[i]);
  }

  CUDA_CHECK(hipFree(d_send));
  CUDA_CHECK(hipFree(d_recv));
  free(h_send);
  free(h_recv);
  cuMPI_Finalize();
  return 0;
}
