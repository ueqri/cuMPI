#include "cuMPI_runtime.h"

static uint64_t getHostHash(const char *string) {
  // based on DJB2, result = result * 33 + char
  uint64_t result = 5381;
  for (int c = 0; string[c] != '\0'; c++) {
    result = ((result << 5) + result) + string[c];
  }
  return result;
}

static void getHostName(char *hostname, int maxlen) {
  gethostname(hostname, maxlen);
  for (int i = 0; i < maxlen; i++) {
    if (hostname[i] == '.') {
      hostname[i] = '\0';
      return;
    }
  }
}

int cuMPI_AllocateOneGPUPerProcess() {
  // TODO
  cuMPI_Init(NULL, NULL);
  return 0;
}

int cuMPI_Initialized(int *flag) { 
  return MPI_Initialized(flag);
}

int cuMPI_Init(int *argc, char ***argv) {

  // initializing MPI
  MPI_CHECK(MPI_Init(argc, argv));
  MPI_CHECK(MPI_Comm_rank(MPI_COMM_WORLD, &myRank));
  MPI_CHECK(MPI_Comm_size(MPI_COMM_WORLD, &nRanks));

  // calculating localRank based on hostname which is used in selecting a GPU
  // localRank -> deviceID
  // myRank    -> NCCL comm rank, myRank will bind to localRank(deviceID) in each node
  getHostName(hostname, 1024);
  hostHashs[myRank] = getHostHash(hostname);
  MPI_CHECK(MPI_Allgather(MPI_IN_PLACE, 0, MPI_DATATYPE_NULL, hostHashs,
                          sizeof(uint64_t), MPI_BYTE, MPI_COMM_WORLD));
  for (int p = 0; p < nRanks; p++) {
    if (p == myRank) {
      break;
    }
    if (hostHashs[p] == hostHashs[myRank]) {
      localRank++;
    }
  }

  // get NCCL **default** unique ID at rank 0 and broadcast it to all others
  if (myRank == 0) {
    ncclGetUniqueId(&id);
  }
  MPI_CHECK(MPI_Bcast((void *)&id, sizeof(id), MPI_BYTE, 0, MPI_COMM_WORLD));

  // picking a GPU based on localRank, allocate device buffers
  printf("Picking Device in %s: %d for MPI Rank: %d/%d\n", hostname, localRank, myRank, nRanks);
  CUDA_CHECK(hipSetDevice(localRank));
  CUDA_CHECK(hipStreamCreate(&commStream));
  defaultCommStream = commStream;

  printf("[%d]: from default stream %p\n", localRank, (void*)commStream);

  // initializing **default** NCCL Communicator
  NCCL_CHECK(ncclCommInitRank(&comm, nRanks, id, myRank));
  // test the initiation
  int tempRank;
  NCCL_CHECK(ncclCommUserRank(comm, &tempRank));
  assert( tempRank == myRank );
  defaultComm = comm;

  // bind comm with stream
  comm2stream[comm] = commStream;

  printf("Initiated NCCL for MPI Rank: %d/%d\n", myRank, nRanks);

  return 0;
}

int cuMPI_Finalize(){
  NCCL_CHECK(ncclCommDestroy(comm));  // finalizing NCCL
  MPI_CHECK(MPI_Finalize());          // finalizing MPI
  printf("[MPI Rank %d] Success.\n", myRank);
  return 0;
}

int cuMPI_Allreduce(const void *sendbuf, void *recvbuf, int count,
  cuMPI_Datatype datatype, cuMPI_Op op, cuMPI_Comm comm){
  #if CUMPI_DEBUG > 1
  printf("AllReduce Communicating...\n");
  #endif
  NCCL_CHECK(ncclAllReduce((const void *)sendbuf, (void *)recvbuf, count,
                            datatype, op, comm, commStream));

  return 0;
}

int cuMPI_Sendrecv(const void *sendbuf, int sendcount, cuMPI_Datatype sendtype,
  int dest, int sendtag,
  void *recvbuf, int recvcount, cuMPI_Datatype recvtype,
  int source, int recvtag,
  cuMPI_Comm comm, cuMPI_Status *status) {
  assert(sendtag == recvtag);
  //(void*)(status), (void)(source); // variable not use

  #if CUMPI_DEBUG > 1
  printf("Send&Receive Communicating...\n");
  #endif
  // peer rank id is `dest`
  NCCL_CHECK(ncclGroupStart());
  NCCL_CHECK(ncclSend(sendbuf, sendcount, sendtype, dest, comm, commStream));
  NCCL_CHECK(ncclRecv(recvbuf, recvcount, recvtype, dest, comm, commStream));
  NCCL_CHECK(ncclGroupEnd());

  return 0;
}


int cuMPI_Complex_Sendrecv(const void *sendreal, const void *sendimag, int sendcount, cuMPI_Datatype sendtype,
  int dest, int sendtag,
  void *recvreal, void *recvimag, int recvcount, cuMPI_Datatype recvtype,
  int source, int recvtag,
  cuMPI_Comm comm, cuMPI_Status *status) {
  
  assert(sendtag == recvtag);

  #if CUMPI_DEBUG > 1
  printf("Send&Receive Complex Communicating...\n");
  #endif
  // peer rank id is `dest`
  NCCL_CHECK(ncclGroupStart());
  NCCL_CHECK(ncclSend(sendreal, sendcount, sendtype, dest, comm, commStream));
  NCCL_CHECK(ncclSend(sendimag, sendcount, sendtype, dest, comm, commStream));
  NCCL_CHECK(ncclRecv(recvreal, recvcount, recvtype, dest, comm, commStream));
  NCCL_CHECK(ncclRecv(recvimag, recvcount, recvtype, dest, comm, commStream));
  NCCL_CHECK(ncclGroupEnd());

  return 0;
}

int cuMPI_Bcast( void *buffer, int count, cuMPI_Datatype datatype, int root, 
  cuMPI_Comm comm ) {
  #if CUMPI_DEBUG > 1
  printf("Bcast Communicating...\n");
  #endif
  // Legacy in-place version of ncclBroadcast in a similar fashion to MPI_Bcast
  NCCL_CHECK(ncclBcast(buffer, count, datatype, root, comm, commStream));

  return 0;
}

int cuMPI_Barrier( cuMPI_Comm comm ) {
  // TODO
  #if CUMPI_DEBUG > 1
  printf("Barrier Waiting...\n");
  #endif
  // ncclCommGetAsyncError
  CUDA_CHECK(hipStreamSynchronize(commStream));
  return 0;
}

int cuMPI_Comm_size(cuMPI_Comm comm, int *size) {
  NCCL_CHECK(ncclCommCount(comm, size));
  #if CUMPI_DEBUG > 1
  printf("Comm Size: [%d]\n", *size);
  #endif
  return 0;
}

int cuMPI_Comm_rank(cuMPI_Comm comm, int *rank) {
  NCCL_CHECK(ncclCommUserRank(comm, rank));
  #if CUMPI_DEBUG > 1
  printf("Comm User Rank: [%d]\n", *rank);
  #endif
  return 0;
}

int cuMPI_NewGlobalComm(cuMPI_Comm *newcomm) {
  ncclUniqueId newid;
  
  // get NCCL new unique ID at rank 0 and broadcast it to all others
  if (myRank == 0) {
    ncclGetUniqueId(&newid);
  }
  MPI_CHECK(MPI_Bcast((void *)&newid, sizeof(newid), MPI_BYTE, 0, MPI_COMM_WORLD));

  // initializing NCCL
  NCCL_CHECK(ncclCommInitRank(newcomm, nRanks, newid, myRank));

  // test the initiation
  int tempRank;
  NCCL_CHECK(ncclCommUserRank(*newcomm, &tempRank));
  assert( tempRank == myRank );

  // create the stream, and bind it to comm
  hipStream_t tmpStream;
  CUDA_CHECK(hipStreamCreate(&tmpStream));
  comm2stream[*newcomm] = tmpStream;

  return 0;
}

int cuMPI_NewPipe(cuMPI_Comm *pipe) {
  // TODO
  return 0;
}

int cuMPI_CocurrentStart(cuMPI_Comm targetcomm) {
  defaultComm = comm;
  defaultCommStream = commStream;
  auto iter = comm2stream.find(targetcomm);
  assert( iter != comm2stream.end() );
  // TODO comm
  commStream = iter->second;
  return 0;
}

int cuMPI_CocurrentEnd(cuMPI_Comm targetcomm) {
  // TODO targetcomm
  comm = defaultComm;
  commStream = defaultCommStream;
  return 0;
}
